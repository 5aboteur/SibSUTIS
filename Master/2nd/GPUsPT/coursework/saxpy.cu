#include "hip/hip_runtime.h"
/*
 * nvcc -std=c++11 saxpy.cu -lcublas
 */

#include <thrust/transform.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>

#include <hipblas.h>
#include <functional>
#include <iostream>
#include <iomanip>
#include <iterator>
#include <algorithm>
#include <chrono>
#include <fstream>
#include <cstdio>
#include <cstdlib>

#define CUDA_ERROR_CHECK(value) {                                  \
        hipError_t _m_cudaStat = value;                          \
        if (_m_cudaStat != hipSuccess) {                         \
                fprintf(stderr, "Error %s at line %d in file %s\n",\
                hipGetErrorString(_m_cudaStat),                  \
                      __LINE__, __FILE__);                        \
                exit(1);                                          \
        }                                                         \
}

void fill(int N, float x_val, float y_val, float *x, float *y)
{
        for (int i = 0; i < N; ++i) { x[i] = x_val; y[i] = y_val; }
}

float test_thrust_saxpy(int N, float a, float x_val, float y_val);
float test_cublas_saxpy(int N, float a, float x_val, float y_val);
float test_native_saxpy(int N, float a, float x_val, float y_val);

/*** DEVICE ***/

__global__ void native_saxpy(int N, float a, float *x, float *y)
{
        int i = blockDim.x * blockIdx.x + threadIdx.x;
        if (i < N) { y[i] = a * x[i] + y[i]; }
}

/*** HOST ***/

int main(void)
{
        std::ofstream fout("result.txt", std::ios_base::app);

        for (int i = 10; i < 28; ++i)
        {
                int N = 1 << i;
                std::cout << N;
                fout << test_native_saxpy(N, 2.0f, 1.0f, 2.0f) << " ";
           fout << test_cublas_saxpy(N, 2.0f, 1.0f, 2.0f) << " ";
           fout << test_thrust_saxpy(N, 2.0f, 1.0f, 2.0f) << " ";
                fout << std::endl;
        }

        return 0;
}

float test_native_saxpy(int N, float a, float x_val, float y_val)
{
        float *h_x, *h_y;
        float *d_x, *d_y;

        float result = a * x_val + y_val;
        int size = N * sizeof(float);

        h_x = (float *)malloc(size);
        h_y = (float *)malloc(size);

        fill(N, x_val, y_val, h_x, h_y);

        CUDA_ERROR_CHECK(hipMalloc(&d_x, size));
        CUDA_ERROR_CHECK(hipMalloc(&d_y, size));

        auto t1 = std::chrono::high_resolution_clock::now();

        CUDA_ERROR_CHECK(hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice));
        CUDA_ERROR_CHECK(hipMemcpy(d_y, h_y, size, hipMemcpyHostToDevice));

        native_saxpy<<<(N + 255) / 256, 256>>>(N, a, d_x, d_y);

        hipDeviceSynchronize();

        CUDA_ERROR_CHECK(hipMemcpy(h_y, d_y, size, hipMemcpyDeviceToHost));

        auto t2 = std::chrono::high_resolution_clock::now();
        auto diff_time = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1);

        float maxError = 0.0f;
        for (int i = 0; i < N; ++i)
        {
                maxError = max(maxError, abs(h_y[i] - 4.0f));
        }

        printf("\nMax error: %f\n", maxError);

        CUDA_ERROR_CHECK(hipFree(d_x));
        CUDA_ERROR_CHECK(hipFree(d_y));

        free(h_x);
        free(h_y);

        return diff_time.count();
}

/*** cuBLAS ***/

float test_cublas_saxpy(int N, float a, float x_val, float y_val)
{
        hipblasHandle_t handle;

        float *h_x, *h_y;
        float *d_x, *d_y;

        float result = a * x_val + y_val;
        int size = N * sizeof(float);

        h_x = (float *)malloc(size);
        h_y = (float *)malloc(size);

        fill(N, x_val, y_val, h_x, h_y);

        CUDA_ERROR_CHECK(hipMalloc(&d_x, size));
        CUDA_ERROR_CHECK(hipMalloc(&d_y, size));

        hipblasCreate(&handle);

        auto t1 = std::chrono::high_resolution_clock::now();

        hipblasSetVector(N, sizeof(h_x[0]), h_x, 1, d_x, 1);
        hipblasSetVector(N, sizeof(h_y[0]), h_y, 1, d_y, 1);
        hipblasSaxpy(handle, N, &a, d_x, 1, d_y, 1);
        hipblasGetVector(N, sizeof(h_y[0]), d_y, 1, h_y, 1);

        auto t2 = std::chrono::high_resolution_clock::now();
        auto diff_time = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1);

        assert(std::all_of(h_y, h_y + N, [&](float value) { return value == result; }));

        hipblasDestroy(handle);

        CUDA_ERROR_CHECK(hipFree(d_x));
        CUDA_ERROR_CHECK(hipFree(d_y));

       free(h_x);
        free(h_y);

        return diff_time.count();
}

/*** THRUST ***/

struct saxpy_functor : public thrust::binary_function<float, float, float>
{
        const float a;
        saxpy_functor(float _a) : a(_a) {}

        __host__ __device__ float operator()(const float &x, const float &y) const
        {
                return a * x + y;
        }
};

float test_thrust_saxpy(int N, float a, float x_val, float y_val)
{
        float result = a * x_val + y_val;

        thrust::host_vector<float> h_x(N), h_y(N);
        thrust::fill(h_x.begin(), h_x.end(), x_val);
        thrust::fill(h_y.begin(), h_y.end(), y_val);

        auto t1 = std::chrono::high_resolution_clock::now();
        thrust::device_vector<float> d_x = h_x;
        thrust::device_vector<float> d_y = h_y;
        thrust::transform(d_x.begin(), d_x.end(), d_y.begin(), d_y.begin(), saxpy_functor(a));

        h_y = d_y;

        auto t2 = std::chrono::high_resolution_clock::now();
        auto diff_time = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1);
        assert(std::all_of(h_y.cbegin(), h_y.cend(), [&](float value) { return value == result; }));

        return diff_time.count();
}
